#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N)
       result[index] = alpha * x[index] + y[index];
}

void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // TODO: allocate device memory buffers on the GPU using
    // hipMalloc.  The started code issues warnings on build because
    // these buffers are used in the call to saxpy_kernel below
    // without being initialized.
    //
    hipMalloc((void **)&device_x, totalBytes/3);
    hipMalloc((void **)&device_y, totalBytes/3);
    hipMalloc((void **)&device_result, totalBytes/3);

    // start timing after allocation of device memory.
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO: copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, totalBytes/3, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, totalBytes/3, hipMemcpyHostToDevice);


    //
    // TODO: insert time here to begin timing only the kernel
    //
    double startKernelTime = CycleTimer::currentSeconds();

    // run saxpy_kernel on the GPU
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);

    //
    // TODO: insert timer here to time only the kernel.  Since the
    // kernel will run asynchronously with the calling CPU thread, you
    // need to call hipDeviceSynchronize() before your timer to
    // ensure the kernel running on the GPU has completed.  (Otherwise
    // you will incorrectly observe that almost no time elapses!)
    //
    hipDeviceSynchronize();
    double endKernelTime = CycleTimer::currentSeconds();


    //
    // TODO: copy result from GPU using hipMemcpy
    //
    hipMemcpy(device_result, resultarray, totalBytes/3, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory.
    // The time elapsed between startTime and endTime is the total
    // time to copy data to the GPU, run the kernel, and copy the
    // result back to the CPU
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallComputeDuration = endKernelTime - startKernelTime;
    printf("Comput time: %.3f ms\n", 1000.f * overallComputeDuration);
    printf("Overall time: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    //
    // TODO free memory buffers on the GPU
    //
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
